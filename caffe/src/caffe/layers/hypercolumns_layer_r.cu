#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: hypercolumns_layer.cpp
	> Author: Jiang Qinhong
	> Mail: mylivejiang@gmail.com
	> Created Time: 2016年07月31日 星期一 19时40分04秒
 ************************************************************************/
#include <vector>
#include <map>
#include <cmath>
#include <algorithm>

#include "caffe/layers/hypercolumns_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "../../../../../../../../../usr/include/c++/4.7/vector"

namespace caffe {

template <typename Dtype>
__global__ void ForwardNormal(const int nthreads,
    const Dtype* bottom_normal, const int num, const int channels,
    const int height, const int width, const int sample_pernum,
    const int* sampling_list, Dtype* const top_normal) {
    // forward top_normals
    CUDA_KERNEL_LOOP(index, nthreads) {
      const int top_n = index / channels; // the top n
      const int bottom_n = top_n / sample_pernum; // the bottom n
      const int c = index % channels; // the same channel of top and bottom
      const int bottom_index = sampling_list[top_n]; // the corresponding index of the bottom
      const Dtype* const bottom_slice = bottom_normal + (bottom_n * channels + c) * height * width;
      top_normal[index] = bottom_slice[bottom_index];
    }
}

template <typename Dtype>
__global__ void ForwardHypercolumns(const int nthreads,
    const int bottom_count, const Dtype** bottom_datas, const int* bottom_channels,
    const int* bottom_heights, const int* bottom_widths, const int sample_pernum,
    const int top_channels,  const int* sampling_list,
    const int W, Dtype* const top_data) {
    //forward hypercolumns, separate for each bottom
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int top_n = index / top_channels; // find the corresponding index in the sampling list
        int offset_channels = index % top_channels;
        int bottom_id = 0;
        while(bottom_id<bottom_count) {
            if(offset_channels - bottom_channels[bottom_id] < 0) {
                break;
            }
            offset_channels -= bottom_channels[bottom_id];
            ++bottom_id;
        }
        // get the corresponding bottom and it's channels

    }
}

template <typename Dtype>
void HyperColumnsLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top) {
    // check and instance the cuda needed data
    if (!cuda_instanced_) {
        instance_cuda_data();
    }

    // generate the sampling list and copy it
    generate_list(bottom[0], false);
    CUDA_CHECK(hipMemcpy(cuda_samplelist_, &selected_points_[0], selected_points_.size()* sizeof(int)));

    // forward step, forward normal first
    Dtype* top_normal = top[1]->mutable_gpu_data();
    const Dtype* bottom_normal = bottom[0]->gpu_data();
    const int count1 = top[1]->count();
    ForwardNormal<Dtype><<<CAFFE_GET_BLOCKS(count1), CAFFE_CUDA_NUM_THREADS>>>(
      count1, bottom_normal, N_, K_, H_, W_, sample_num_, cuda_samplelist_, top_normal
    );

    // then forward the hypercolumns
    Dtype* top_hypercolumns = top[0]->mutable_gpu_data();
    vector<const Dtype*> bottom_datas;
    const int bottom_count = bottom.size() - 1;
    for (int i = 1; i < bottom.size(); ++i) {
        bottom_datas.push_back(bottom[i]->gpu_data());
    }
    const int nthreads = N_ * sample_num_ * total_channels_;
    ForwardHypercolumns<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_count, &bottom_datas[0], cuda_channels_, cuda_heights_, cuda_widths_, sample_num_,
        total_channels_, cuda_samplelist_, W_, top_hypercolumns
    );
}

template <typename Dtype>
__global__ void BackwardHypercolumns(const int nthreads,
     const Dtype** bottom_diffs, const int* bottom_channels,
     const int* bottom_heights, const int* bottom_widths, const int sample_pernum,
     const int top_channels,  const int* sampling_list,
     const int W, Dtype* const top_diff) {
  // backward hypercolumns, seperate for each bottom
    CUDA_KERNEL_LOOP(index, nthreads) {

    }
}

template <typename Dtype>
void HyperColumnsLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
   // backward step, back the diff in top[0] to the bottom, except bottom[0]
    const Dtype* top_diff = top[0]->gpu_diff();
    vector<Dtype*> bottom_diffs;
    for (int i = 1; i < bottom.size(); ++i) {
        bottom_diffs.push_back(bottom[i]->mutable_gpu_diff());
    }
    const int nthreads = N_ * sample_num_ * total_channels_;
    BackwardHypercolumns<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, &bottom_diffs[0], cuda_channels_, cuda_heights_, cuda_widths_, sample_num_,
        total_channels_, cuda_samplelist_, W_, top_diff
    );
}

template <typename Dtype>
void HyperColumnsLayer<Dtype>::instance_cuda_data() {
    // instance the sampling list
    CUDA_CHECK(hipMalloc(&cuda_samplelist_, selected_points_.size() * sizeof(int)));
    // instance the width, height and channel
    CUDA_CHECK(hipMalloc(&cuda_widths_, width_.size() * sizeof(int)));
    CUDA_CHECK(hipMemcpy(cuda_widths_, &width_[0], width_.size()* sizeof(int)));
    CUDA_CHECK(hipMalloc(&cuda_heights_, height_.size() * sizeof(int)));
    CUDA_CHECK(hipMemcpy(cuda_heights_, &height_[0], height_.size()* sizeof(int)));
    CUDA_CHECK(hipMalloc(&cuda_channels_, channels_.size() * sizeof(int)));
    CUDA_CHECK(hipMemcpy(cuda_channels_, &channels_[0], channels_.size()* sizeof(int)));
    generate_bilinear_map(); // generate the mapping list
    CUDA_POST_KERNEL_CHECK;
    cuda_instanced_ = true;
}

template <typename Dtype>
HyperColumnsLayer<Dtype>::~HyperColumnsLayer() {
    CUDA_CHECK(hipFree(cuda_samplelist_));
    CUDA_CHECK(hipFree(cuda_widths_));
    CUDA_CHECK(hipFree(cuda_heights_));
    CUDA_CHECK(hipFree(cuda_channels_));
    CUDA_CHECK(hipFree(cuda_map_lists_));
    CUDA_POST_KERNEL_CHECK;
    cuda_instanced_ = false;
}

template <typename Dtype>
void HyperColumnsLayer<Dtype>::generate_bilinear_map() {
// generate the bilinear map all in one at begin
    const int total_index = H_ * W_;
    const int bottom_count = width_.size() - 1;
    CUDA_CHECK(hipMalloc(&cuda_map_lists_, 6 * bottom_count * total_index * sizeof(double)));
    // get the value for every sample index
    int h, w;
    double fw, fh, cw, ch;
    double tempw, temph;
    int count = 0;
    for (int index = 0; index < total_index; ++index) {
        h = index / W_;
        w = index % W_;
        for (int b = 1; b < width_.size(); ++b) {
            tempw = (w - padf_[b]) / scalef_[b];
            temph = (h - padf_[b]) / scaled_[b];
            fw = static_cast<int>(floor(tempw));
            fh = static_cast<int>(floor(temph));
            cw = static_cast<int>(ceil(tempw));
            ch = static_cast<int>(ceil(temph));
            // boundary check
            fw = fw > 0 ? fw : 0;
            cw = cw > 0 ? cw : 0;
            fh = fh > 0 ? fh : 0;
            ch = ch > 0 ? ch : 0;
            cw = cw < width_[b] ? cw : fw;
            ch = ch < height_[b] ? ch : fh;
            cuda_map_lists_[count++] = tempw;
            cuda_map_lists_[count++] = temph;
            cuda_map_lists_[count++] = fw;
            cuda_map_lists_[count++] = fh;
            cuda_map_lists_[count++] = cw;
            cuda_map_lists_[count++] = ch;
        }
    }

}




INSTANTIATE_LAYER_GPU_FUNCS(HyperColumnsLayer);
}// namespace caffe
